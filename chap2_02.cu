#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 256

__global__ void matrix_vector_multi_gpu_1_1(float *A_d, float *B_d, float *C_d) {
    int i, j;

    for (j=0;j<N;j++) {
        A_d[j]=0.0;
        for(i=0;i<N;i++) {
            A_d[j]=A_d[j]+B_d[j*N+i]*C_d[i];
        }
    }
}

int main() {

    int i, j;
    float A[N], B[N*N], C[N];   //ホスト
    float *A_d, *B_d, *C_d;     //デバイス

    dim3 blocks(1,1,1);         //blockの配置
    dim3 threads(1,1,1);        //threadの配置

    for(j=0; j<N; j++) {
        for (i=0; i<N; i++) {
            B[j*N+i] = ((float)j)/256.0;
        }
    }

    for(j=0; j<N; j++) {
        C[j] = 1.0F;
    }

    hipMalloc((void**)&A_d, N*sizeof(float));      //デバイス側のメモリの確保
    hipMalloc((void**)&B_d, N*N*sizeof(float));
    hipMalloc((void**)&C_d, N*sizeof(float));

    hipMemcpy(A_d, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, N*sizeof(float), hipMemcpyHostToDevice);

    matrix_vector_multi_gpu_1_1<<< blocks, threads >>>(A_d, B_d, C_d);

    hipMemcpy(A, A_d, N*sizeof(float), hipMemcpyDeviceToHost);

    for(j=0;j<N;j++) {
        printf("A[ %d ]=%f \n", j, A[j]);
    }

    hipFree(A_d); //メモリの開放
    hipFree(B_d);
    hipFree(C_d);

}